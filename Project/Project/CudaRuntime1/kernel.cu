﻿#include "hip/hip_runtime.h"
#include "thrust/device_vector.h"
#include <thrust/random.h>
#include <thrust/transform.h>

#include <chrono>
#include <iostream>

struct GenRand {
    __device__
        int operator () (int idx) {
        thrust::default_random_engine randEng;
        thrust::uniform_int_distribution<int> uniDist;
        randEng.discard(idx);
        return uniDist(randEng);
    }
};

int main() {
    const size_t N = 3e4;
    const size_t size = N * N;
    std::cout << "elements: " << size << '\n';

    thrust::device_vector<int> a(size);
    thrust::device_vector<int> b(size);
    thrust::device_vector<int> c(size);

    auto start = std::chrono::high_resolution_clock::now();
    thrust::transform(
        thrust::make_counting_iterator(0ULL),
        thrust::make_counting_iterator(size),
        a.begin(),
        GenRand());
    thrust::transform(
        thrust::make_counting_iterator(0ULL),
        thrust::make_counting_iterator(size),
        b.begin(),
        GenRand());
    auto end = std::chrono::high_resolution_clock::now();
    auto end = std::chrono::high_resolution_clock::now();
    std::cout << std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count() << std::endl;
    return 0;
}
