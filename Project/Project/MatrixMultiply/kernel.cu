#include "hip/hip_runtime.h"
﻿#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/random.h>
#include <iostream>
#include <fstream>
#include <chrono>
#include <string>
#include <sstream>
#include <random>

struct GenRand {
private:
    int seed;

public:
    GenRand(int seed)
        : seed(seed) {
    }

    __device__ float operator()(int idx) {
        thrust::default_random_engine randEng(seed);
        thrust::uniform_int_distribution<int> uniDist(1, 10);
        randEng.discard(idx);
        return uniDist(randEng);
    }
};

__global__ void matrixMultiplicationKernel(const float* A, const float* B, float* C, size_t N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < N; ++k) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

void matrixMultiplication(const thrust::device_vector<float>& A, const thrust::device_vector<float>& B,
                          thrust::device_vector<float>& C, int N, bool mp) {
    dim3 threadsPerBlock, blocksPerGrid;
    if (mp) {
        threadsPerBlock = dim3(16, 16, 1);
        blocksPerGrid =
            dim3((N + threadsPerBlock.x - 1) / threadsPerBlock.x, (N + threadsPerBlock.y - 1) / threadsPerBlock.y);
    } else {
        threadsPerBlock = dim3(1, 1, 1);
        blocksPerGrid =
            dim3((N + threadsPerBlock.x - 1) / threadsPerBlock.x, (N + threadsPerBlock.y - 1) / threadsPerBlock.y);
    }

    matrixMultiplicationKernel<<<blocksPerGrid, threadsPerBlock>>>(
        thrust::raw_pointer_cast(A.data()), thrust::raw_pointer_cast(B.data()), thrust::raw_pointer_cast(C.data()), N);
}

int main(int argc, const char** argv) {
    if (argc != 6) {
        std::cout << "You should specify start size, end size, step size of matrix and output filename and use "
                     "parallel or not!"
                  << std::endl;
        return -1;
    }

    srand(time(0));
    std::ostringstream output;
    output << "count,allocation_time,fill_time,calculation_time\n";

    for (size_t N = std::atoi(argv[1]); N < std::atoi(argv[2]); N += std::atoi(argv[3])) {
        auto start = std::chrono::system_clock::now();
        thrust::device_vector<float> A(N * N);
        thrust::device_vector<float> B(N * N);
        thrust::device_vector<float> C(N * N);
        auto end = std::chrono::system_clock::now();
        auto elapsed = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();

        output << N << ',' << elapsed << ',';

        start = std::chrono::system_clock::now();
        thrust::transform(thrust::make_counting_iterator(0ULL), thrust::make_counting_iterator(N * N), A.begin(),
                          GenRand(rand()));
        thrust::transform(thrust::make_counting_iterator(0ULL), thrust::make_counting_iterator(N * N), B.begin(),
                          GenRand(rand()));
        end = std::chrono::system_clock::now();
        elapsed = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();

        output << elapsed << ',';

        bool use_mp = std::atoi(argv[5]);
        start = std::chrono::system_clock::now();
        matrixMultiplication(A, B, C, N, use_mp);
        hipDeviceSynchronize();
        end = std::chrono::system_clock::now();
        elapsed = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();

        output << elapsed << '\n';
    }
    std::ofstream output_file(std::string(argv[4]) + ".csv");
    output_file << output.str();

    return 0;
}
