﻿#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/random.h>
#include <iostream>

struct GenRand {
private:
    int seed;

public:
    GenRand(int seed)
        : seed(seed) {
    }

    __device__ float operator()(int idx) {
        thrust::default_random_engine randEng(seed);
        thrust::uniform_int_distribution<int> uniDist(1, 10);
        randEng.discard(idx);
        return uniDist(randEng);
    }
};

int main(int argc, const char** argv) {
    if (argc != 2) {
        std::cout << "Enter how much bytes to allocate!" << std::endl;
    }
    size_t count = std::atoll(argv[1]);
    thrust::device_vector<std::uint8_t> garbage(count, 0);
    thrust::transform(thrust::make_counting_iterator(0ULL), thrust::make_counting_iterator(garbage.size()),
                      garbage.begin(), GenRand(0));
    std::cout << "Allocated " << count << " bytes" << std::endl;
    std::string input;
    std::cin >> input;
    return 0;
}
